#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>
#include "utils.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabsf(y), fabsf(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}


// morton utils
inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
    uint32_t xx = __expand_bits(x);
    uint32_t yy = __expand_bits(y);
    uint32_t zz = __expand_bits(z);
    return xx | (yy << 1) | (zz << 2);
}

__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}
void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(
        coords.data_ptr<int>(), N, 
        indices.data_ptr<int>()
    );
}


inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
    x = x & 0x49249249;
    x = (x | (x >> 2)) & 0xc30c30c3;
    x = (x | (x >> 4)) & 0x0f00f00f;
    x = (x | (x >> 8)) & 0xff0000ff;
    x = (x | (x >> 16)) & 0x0000ffff;
    return x;
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}

void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}






// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    // const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}






// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, ts: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps

template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float bound, const bool contract,//通过bound确定块的尺度
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * ts,//这里是需要记录下来的contraction点
    int * rays,//[N_rays, 2]
    int * counter,//[1]
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // is first pass running.
    const bool first_pass = (xyzs == nullptr);

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    rays += n * 2;

    uint32_t num_steps = max_steps;

    if (!first_pass) {
        uint32_t point_index = rays[0];

        num_steps = rays[1];

        xyzs += point_index * 3;
        dirs += point_index * 3;
        ts += point_index * 2;
    }

    // ray marching

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;
    
    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    // printf("far \n%f",far);
    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * bound / H;
    // const float dt_max = 1e10f;
    
    float t0 = near;
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;
    float t = t0;
    uint32_t step = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);//bound default to 2
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]
        //C是cascade，
        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // contraction
        float cx = x, cy = y, cz = z;
        const float mag = fmaxf(fabsf(x), fmaxf(fabsf(y), fabsf(z)));
        if (contract && mag > 1) {
            // L-INF norm
            const float Linf_scale = (2 - 1 / mag) / mag;
            cx *= Linf_scale;
            cy *= Linf_scale;
            cz *= Linf_scale;
        }
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (cx * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (cy * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (cz * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        // if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, step);

        if (occ || (contract && mag > 1)) {
            step++;
            t += dt;
            if (!first_pass) {
                xyzs[0] = cx; // write contracted coordinates!
                xyzs[1] = cy;
                xyzs[2] = cz;
                dirs[0] = dx;
                dirs[1] = dy;
                dirs[2] = dz;
                ts[0] = t;
                ts[1] = dt;
                xyzs += 3;
                dirs += 3;
                ts += 2;
            }
        // contraction case: cannot apply voxel skipping.
        // } else if (contract && mag > 1) {
        //     t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - cx) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - cy) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - cz) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                dt = clamp(t * dt_gamma, dt_min, dt_max);
                t += dt;
            } while (t < tt);
        }
    }

    // printf("[n=%d] step=%d, near=%f, far=%f, dt=%f, num_steps=%f\n", n, step, near, far, dt_min, (far - near) / dt_min);

    // write rays
    if (first_pass) {// id, offset, num_steps
        uint32_t point_index = atomicAdd(counter, step);
        rays[0] = point_index;
        rays[1] = step;
    }
}

void march_rays_train(
    const at::Tensor rays_o, 
    const at::Tensor rays_d, 
    const at::Tensor grid, 
    const float bound, 
    const bool contract, 
    const float dt_gamma, 
    const uint32_t max_steps, 
    const uint32_t N, const uint32_t C, 
    const uint32_t H, const at::Tensor nears, 
    const at::Tensor fars, 
    at::optional<at::Tensor> xyzs, 
    at::optional<at::Tensor> dirs, 
    at::optional<at::Tensor> ts, 
    at::Tensor rays, at::Tensor counter, at::Tensor noises
) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, contract, dt_gamma, max_steps, N, C, H, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(),
            xyzs.has_value() ? xyzs.value().data_ptr<scalar_t>() : nullptr,
            dirs.has_value() ? dirs.value().data_ptr<scalar_t>() : nullptr,
            ts.has_value() ? ts.value().data_ptr<scalar_t>() : nullptr,
            rays.data_ptr<int>(), counter.data_ptr<int>(), noises.data_ptr<scalar_t>());
    }));
}
