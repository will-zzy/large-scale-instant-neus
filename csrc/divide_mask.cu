#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <iostream>
#include <ATen/ATen.h>
#include <math.h>
#include <ATen/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "hip/hip_vector_types.h"
using namespace std;



__global__ void packbits_u32_kernel(
    torch::PackedTensorAccessor32<int32_t,1,torch::RestrictPtrTraits> idx_array,
    torch::PackedTensorAccessor64<int64_t,1,torch::RestrictPtrTraits> bits_array
){
    // const int32_t n = blockIdx.x * blockDim.x + threadIdx.x;//一维时
    const int32_t n = blockIdx.x*blockDim.x*blockDim.y + threadIdx.x*blockDim.y + threadIdx.y;//二维时
    if(n > bits_array.size(0))
        return;
    int mask_size = 32;
    if (n == bits_array.size(0))
        mask_size = (idx_array.size(0) % 32) - 1;
    const int64_t flag = 1;
    for(int i = 0 ; i < mask_size ; i++){
        int32_t hit_pix = idx_array[n*32 + i];
        if (hit_pix > 0){
            bits_array[n] |= flag << i;
        }
    }
}


torch::Tensor packbits_u32(
    torch::Tensor idx_array,
    torch::Tensor bits_array
){
    // 每个线程处理32位长数据即32个像素
    const int num_pixs = std::ceil(idx_array.size(0)/32);
    // const int threads = 256, blocks = (num_pixs+threads-1)/threads;
    const int BLOCK_W = 64;
    const int BLOCK_H = 16;
    const dim3 blockSize(BLOCK_W,BLOCK_H,1);
    const dim3 gridSize((num_pixs + BLOCK_W*BLOCK_H - 1)/(BLOCK_W * BLOCK_H),1,1);
    // const dim3 gridSize(8,1,1);

    // torch::Tensor bit_array = torch::zeros({bits_array.size(0)},bits_array.options());
    AT_DISPATCH_ALL_TYPES(idx_array.type(),"packbits_u32",
    // AT_DISPATCH_ALL_TYPES(idx_array.type(),"packbits_u64_cu",
    ([&] {
        packbits_u32_kernel<<<gridSize, blockSize>>>(
        // packbits_u64_kernel<<<4, 64>>>(
            idx_array.packed_accessor32<int32_t,1,torch::RestrictPtrTraits>(),
            bits_array.packed_accessor64<int64_t,1,torch::RestrictPtrTraits>()
        );
    }));
    return bits_array;
}

__global__ void un_packbits_u32_kernel(
    torch::PackedTensorAccessor32<int32_t,1,torch::RestrictPtrTraits> idx_array,
    torch::PackedTensorAccessor64<int64_t,1,torch::RestrictPtrTraits> bits_array
){
    // const int32_t n = blockIdx.x * blockDim.x + threadIdx.x;//一维时
    const int32_t n = blockIdx.x*blockDim.x*blockDim.y + threadIdx.x*blockDim.y + threadIdx.y;//二维时

    if(n > bits_array.size(0))
        return;
    int mask_size = 32;
    if (n == bits_array.size(0))
        mask_size = (idx_array.size(0) % 32) - 1;
    const int64_t flag = 1;
    for(int i = 0 ; i < mask_size ; i++){
        if (bits_array[n] & (flag << i)){
            idx_array[n*32 + i]++;
        }
    }
}
torch::Tensor un_packbits_u32(
    torch::Tensor idx_array,
    torch::Tensor bits_array
){
    // 每个线程处理64位长数据即64个像素
    const int num_pixs = std::ceil(idx_array.size(0)/64);
    // const int threads = 256, blocks = (num_pixs+threads-1)/threads;
    const int BLOCK_W = 64;
    const int BLOCK_H = 16;
    const dim3 blockSize(BLOCK_W,BLOCK_H,1);
    const dim3 gridSize((num_pixs + BLOCK_W*BLOCK_H - 1)/(BLOCK_W * BLOCK_H),1,1);

    AT_DISPATCH_ALL_TYPES(idx_array.type(),"un_packbits_u32",
    ([&] {
        un_packbits_u32_kernel<<<gridSize, blockSize>>>(
            idx_array.packed_accessor32<int32_t,1,torch::RestrictPtrTraits>(),
            bits_array.packed_accessor64<int64_t,1,torch::RestrictPtrTraits>()
        );
    }));
    return idx_array;
}




__global__ void distance_mask_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dirsMap,//视线方向 [N 3]
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> locMap,//相机光心 [3]
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> centroids,//分块质心 [C 3]
    torch::PackedTensorAccessor32<int32_t,2,torch::RestrictPtrTraits> mask,//一张图像对于C个质心的mask值 [N , C]

    const float threshould//重叠阈值
){
    // const int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    // const int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    // const int n = threadId_2D + (blockDim.x*blockDim.y)*blockId_2D;
    const int32_t n = blockIdx.x*blockDim.x*blockDim.y + threadIdx.x*blockDim.y + threadIdx.y;//二维时
    if(n >= dirsMap.size(0))
        return;
    const float dx = dirsMap[n][0], dy = dirsMap[n][1], dz = dirsMap[n][2];
    const float ox = locMap[0], oy = locMap[1], oz = locMap[2];
    float3 dir = make_float3(dx,dy,dz);
    float3 loc = make_float3(ox,oy,oz);
    float* d = new float[centroids.size(0)];
    float d_min = 99999.9;
    
    dir /= length(dir);

    for(int i  = 0; i < centroids.size(0); i++){
        float3 centroid = make_float3(centroids[i][0],centroids[i][1],centroids[i][2]); 
        float3 l_vec = centroid - loc;
        float3 d_vec = cross(l_vec,dir);
        d[i] = length(d_vec);
        if (d_min >= d[i])
            d_min = d[i];

        // \  d   |
        //  *-----|
        //   \    |
        // l  \   | dir   d = |(l X dir)|/|dir|
        //     \  |
        //      \ |
        //       \|

    }
    
    for(int i  = 0; i < centroids.size(0); i++){
        if (d[i] <= (d_min * threshould))
            mask[n][i] = 1;
    }
    delete d;
    d = nullptr;
}


torch::Tensor distance_mask(
    torch::Tensor dirsMap,//[WxH , 3]
    torch::Tensor locMap,//[WxH , 3]
    torch::Tensor centroids,//[C , 3]
    torch::Tensor mask,//[WxH , C]
    const float threshould//重叠阈值
){
    const int num_pisxels = dirsMap.size(0);
    const int BLOCK_W = 64;
    const int BLOCK_H = 16;
    const dim3 blockSize(BLOCK_W,BLOCK_H,1);
    const dim3 gridSize((num_pisxels + BLOCK_W*BLOCK_H - 1)/(BLOCK_W*BLOCK_H),1,1);
    // const dim3 gridSize(1,1,1);
    AT_DISPATCH_ALL_TYPES(dirsMap.type(),"distance_mask",
    ([&] {
        distance_mask_kernel<<<gridSize, blockSize>>>(
        // packbits_u64_kernel<<<4, 64>>>(
            dirsMap.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            locMap.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
            centroids.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            mask.packed_accessor32<int32_t,2,torch::RestrictPtrTraits>(),
            threshould

        );
    }));
    return mask;
}




__global__ void mega_nerf_mask_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dirsMap,//视线方向 [N 3]
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> locMap,//相机光心 [3]
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> centroids,//分块质心 [C 3]
    torch::PackedTensorAccessor32<int32_t,2,torch::RestrictPtrTraits> mask,//一张图像对于C个质心的mask值 [N , C]
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> t_range,//一张图像对于C个质心的mask值 [N , 2]
    
    const int samples,//每条射线上采样点数
    const float threshould//重叠阈值
){
    const int32_t n = blockIdx.x*blockDim.x*blockDim.y + threadIdx.x*blockDim.y + threadIdx.y;//二维时
    if(n >= dirsMap.size(0))
        return;
    const float dx = dirsMap[n][0], dy = dirsMap[n][1], dz = dirsMap[n][2];
    const float ox = locMap[0], oy = locMap[1], oz = locMap[2];
    float3 dir = make_float3(dx,dy,dz);
    float3 loc = make_float3(ox,oy,oz);
    
    dir /= length(dir);



    float3 current_pt = loc + dir * t_range[n][0];
    const float dt = (t_range[n][1] - t_range[n][0]) / samples;
    for(int i = 0; i < samples; i++){//遍历射线上所有采样点
        float d_min = 999.9;
        for(int j = 0; j < centroids.size(0); j++){//先计算离采样点最新的距离值
            float3 centroid = make_float3(centroids[j][0],centroids[j][1],centroids[j][2]);
            float d_tmp = length(current_pt - centroid);
            if (d_min > d_tmp)
                d_min = d_tmp;
        }

        for(int j = 0; j < centroids.size(0); j++){
            float3 centroid = make_float3(centroids[j][0],centroids[j][1],centroids[j][2]);
            float d_ratio_tmp = length(current_pt - centroid)/(d_min+1e-8);
            if(threshould >= d_ratio_tmp){
                mask[n][j]=1;
            }
        }
        current_pt += dt * dir;
    }
}


torch::Tensor mega_nerf_mask(
    torch::Tensor dirsMap,//[WxH , 3]
    torch::Tensor locMap,//[WxH , 3]
    torch::Tensor centroids,//[C , 3]
    torch::Tensor t_range,
    const int samples,//每条射线上采样点数
    const float threshould//重叠阈值
){
    
    const int BLOCK_W = 64;
    const int BLOCK_H = 16;
    const dim3 blockSize(BLOCK_W,BLOCK_H,1);
    const dim3 gridSize((dirsMap.size(0) + BLOCK_W*BLOCK_H - 1)/(BLOCK_W*BLOCK_H),1,1);
    auto mask = torch::zeros({dirsMap.size(0),centroids.size(0)}, 
                                        torch::dtype(torch::kInt32).device(dirsMap.device()));
    AT_DISPATCH_ALL_TYPES(dirsMap.type(),"mega_nerf_mask",
    ([&] {
        mega_nerf_mask_kernel<<<gridSize, blockSize>>>(
            dirsMap.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            locMap.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
            centroids.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            mask.packed_accessor32<int32_t,2,torch::RestrictPtrTraits>(),
            t_range.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            samples,
            threshould
        );
    }));
    return mask;
}
